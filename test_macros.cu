
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
#ifdef CUDA_AVAILABLE
    printf("CUDA_AVAILABLE is defined\n");
#else
    printf("CUDA_AVAILABLE is NOT defined\n");
#endif

#ifdef OPTIX_AVAILABLE
    printf("OPTIX_AVAILABLE is defined\n");
#else
    printf("OPTIX_AVAILABLE is NOT defined\n");
#endif

    return 0;
}
